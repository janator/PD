
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>


#define LOG_NUM_BANKS 5
#define GET_OFFSET(idx) (idx >> LOG_NUM_BANKS)
#define BLOCK_SIZE 256


__global__
void BlockScan(int* in_data, int* out_data, int* sum, int size) {

  extern __shared__ int shared_data[];

  unsigned int tid = threadIdx.x;
  if (tid < size) {
    shared_data[tid + GET_OFFSET(tid)] = in_data[tid];
  } else {
    shared_data[tid + GET_OFFSET(tid)] = 0;
  }


  __syncthreads();

  for (unsigned int shift = 1; shift < blockDim.x; shift <<= 1 ) {
    int ai = shift * (2 * tid + 1) - 1;
    int bi = shift * (2 * tid + 2) - 1;

    if (bi < blockDim.x) {
      shared_data[bi + GET_OFFSET(bi)] += shared_data[ai + GET_OFFSET(ai)];
    }

    __syncthreads();
  }

  if (tid == 0) {
    sum[0] = shared_data[blockDim.x - 1 + GET_OFFSET(blockDim.x - 1)];
    shared_data[blockDim.x - 1 + GET_OFFSET(blockDim.x- 1)] = 0;
  }

  __syncthreads();

  int temp;
  for (unsigned int shift = blockDim.x / 2; shift > 0; shift >>= 1) {
    int bi = shift * (2 * tid + 2) - 1;
    int ai = shift * (2 * tid + 1) - 1;
    int ai_offset = ai + GET_OFFSET(ai);
    int bi_offset = bi + GET_OFFSET(bi);
    if (bi < blockDim.x) {
      temp = shared_data[ai_offset]; // blue in temp

      shared_data[ai_offset] = shared_data[bi_offset]; // orange

      shared_data[bi_offset] = temp + shared_data[bi_offset];
    }
    __syncthreads();

  }
  out_data[tid] = shared_data[tid + GET_OFFSET(tid)];

  __syncthreads();

}

__global__
void AddInScan(int* in_data, int* sum, int size) {
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size && index >= blockIdx.x) {
    in_data[index] += sum[blockIdx.x];
  }
}

__global__
void MakeFlag(int* in_data, int* less_flag, int* equal_flag, int* greater_flag, int size) {
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  int pivot = in_data[size - 1];
  if (index < size) {
    less_flag[index] = (int) (in_data[index] < pivot);
    equal_flag[index] = (int) (in_data[index] == pivot);
    greater_flag[index] = (int) (in_data[index] > pivot);
  }
}

void Scan(int* d_array, int* d_localscan, int size, int* d_full_sum) {
  // сканируем массив поблочно
  int num_blocks = size % BLOCK_SIZE == 0 ? size / BLOCK_SIZE : size / BLOCK_SIZE + 1;
  int* d_sum;
  hipMalloc(&d_sum, sizeof(int) * num_blocks);

  for (int i = 0; i < num_blocks; ++i) {
    int cur_size = BLOCK_SIZE * (i + 1) <= size ? BLOCK_SIZE : size % BLOCK_SIZE;
    BlockScan <<< 1, BLOCK_SIZE, sizeof(int) * (BLOCK_SIZE + GET_OFFSET(BLOCK_SIZE)) >>> (&d_array[i * BLOCK_SIZE],
        &d_localscan[i * BLOCK_SIZE], &d_sum[i], cur_size);
  }

  int* d_sum_out;
  hipMalloc(&d_sum_out, sizeof(int) * (num_blocks + 1));

  // сканируем суммы в конце блоков, если массив меньше 1^256, то должно влезть в один блок
  // также сохраним конечную сумму, пригодится для размера массивов
  BlockScan <<< 1, BLOCK_SIZE, sizeof(int) * (BLOCK_SIZE + GET_OFFSET(BLOCK_SIZE)) >>> (d_sum, d_sum_out, d_full_sum, num_blocks);

  // Добавляем суммы к массиву префикс сумм
  num_blocks = (size + 1) % BLOCK_SIZE == 0 ? (size + 1) / BLOCK_SIZE : (size + 1) / BLOCK_SIZE + 1;
  AddInScan <<<num_blocks, BLOCK_SIZE>>> (d_localscan, d_sum_out, size + 1);
}

__global__
void Split(int *in_data, int* out_data, int* flag, int size) {
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

  // возможно стоит подгрузить в shared_memory flag
  if (index < size - 1 &&  flag[index] < flag[index + 1]) {
    out_data[flag[index]] = in_data[index];
  }
}
__global__
void Copy(int* d_from_array, int* d_to_array, int size) {
  unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size) {
    d_to_array[index] = d_from_array[index];
  }
}

void QuickSort(int *d_array, int* d_splited, int size) {
  int num_blocks = size % BLOCK_SIZE == 0 ? size / BLOCK_SIZE : size / BLOCK_SIZE + 1;
  int* d_less_flag;
  hipMalloc(&d_less_flag, sizeof(int) * size);
  int* d_equal_flag;
  hipMalloc(&d_equal_flag, sizeof(int) * size);
  int* d_greater_flag;
  hipMalloc(&d_greater_flag, sizeof(int) * size);

  // делаем массивы сравнений
  MakeFlag <<<num_blocks, BLOCK_SIZE, 1>>> (d_array, d_less_flag, d_equal_flag, d_greater_flag, size);

  // сканируем эти массивы
  int *d_less_flag_scan;
  hipMalloc(&d_less_flag_scan, sizeof(int) * (size + 1));
  int *d_equal_flag_scan;
  hipMalloc(&d_equal_flag_scan, sizeof(int) * (size + 1));
  int *d_greater_flag_scan;
  hipMalloc(&d_greater_flag_scan, sizeof(int) * (size + 1));
  int* d_less_flag_size;
  int* d_equal_flag_size;
  int* d_greater_flag_size;
  hipMalloc(&d_less_flag_size, sizeof(int));
  hipMalloc(&d_equal_flag_size, sizeof(int));
  hipMalloc(&d_greater_flag_size, sizeof(int));

  Scan(d_less_flag, d_less_flag_scan, size, d_less_flag_size);
  int h_less_flag_size, h_equal_flag_size, h_greater_flag_size;
  hipMemcpy(&h_less_flag_size, d_less_flag_size, sizeof(int), hipMemcpyDeviceToHost);

  Scan(d_equal_flag, d_equal_flag_scan, size, d_equal_flag_size);
  hipMemcpy(&h_equal_flag_size, d_equal_flag_size, sizeof(int), hipMemcpyDeviceToHost);
  Scan(d_greater_flag, d_greater_flag_scan, size, d_greater_flag_size);
  hipMemcpy(&h_greater_flag_size, d_greater_flag_size, sizeof(int), hipMemcpyDeviceToHost);

  // перемещаем в наши новые массивы сначала меньшие значения, потом равные, потом большие
  // !!! последний элемент пивот, надо скопировать тоже !!!


  Split <<<num_blocks, BLOCK_SIZE>>> (d_array, d_splited, d_less_flag_scan, size + 1);
  Split <<<num_blocks, BLOCK_SIZE>>> (d_array, &d_splited[h_less_flag_size], d_equal_flag_scan, size + 1);
  Split <<<num_blocks, BLOCK_SIZE>>> (d_array, &d_splited[h_less_flag_size + h_equal_flag_size], d_greater_flag_scan, size + 1);

  int *d_new_splited_less;
  hipMalloc(&d_new_splited_less, sizeof(int) * h_less_flag_size);

  int *d_new_splited_greater;
  hipMalloc(&d_new_splited_greater, sizeof(int) * h_greater_flag_size);

  if (h_less_flag_size > 1) {
    QuickSort(d_splited, d_new_splited_less, h_less_flag_size);
    int num_blocks = h_less_flag_size % BLOCK_SIZE == 0 ? h_less_flag_size / BLOCK_SIZE : h_less_flag_size / BLOCK_SIZE + 1;
    Copy<<< num_blocks, BLOCK_SIZE>>> (d_new_splited_less, d_splited, h_less_flag_size);
  }
  if (h_greater_flag_size > 1) {
    QuickSort(&d_splited[h_less_flag_size + h_equal_flag_size], d_new_splited_greater, h_greater_flag_size);
    int num_blocks = h_greater_flag_size % BLOCK_SIZE == 0 ? h_greater_flag_size / BLOCK_SIZE : h_greater_flag_size / BLOCK_SIZE + 1;
    Copy<<< num_blocks, BLOCK_SIZE>>> (d_new_splited_greater, &d_splited[h_equal_flag_size + h_less_flag_size], h_greater_flag_size);
  }
}

int partition (int *a, int p, int r)
{
  int x = *(a+r);
  int i = p - 1;
  int j;
  int tmp;
  for (j = p; j < r; j++)
  {
    if (*(a+j) <= x)
    {
      i++;
      tmp = *(a+i);
      *(a+i) = *(a+j);
      *(a+j) = tmp;
    }
  }
  tmp = *(a+r);
  *(a+r) = *(a+i+1);
  *(a+i+1) = tmp;
  return i + 1;
}

void SlowQuicksort (int *a, int p, int r)
{
  int q;
  if (p < r)    {
    q = partition (a, p, r);
    SlowQuicksort (a, p, q-1);
    SlowQuicksort (a, q+1, r);
  }
}

int main() {
  const int block_size = 256;
  hipEvent_t start;
  hipEvent_t stop;

  // Creating event
  hipEventCreate(&start);
  hipEventCreate(&stop);

  const int array_size = 1024;
  int* h_array = new int[array_size];
  for (int i = 0; i < array_size; ++i) {
    h_array[i] = i % 4;
  }
  int* d_array;

  hipMalloc(&d_array, sizeof(int) * array_size);
  hipMemcpy(d_array, h_array, sizeof(int) * array_size, hipMemcpyHostToDevice);

  int* d_sorted;
  hipMalloc(&d_sorted, sizeof(int) * array_size);
  hipEventRecord(start);

  QuickSort(d_array, d_sorted, array_size);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  std::cout << milliseconds << " elapsed fast" << std::endl;
  hipEvent_t start2;
  hipEvent_t stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  hipEventRecord(start2);

  SlowQuicksort(h_array, 0, array_size);

  hipEventRecord(stop2);
  hipEventSynchronize(stop2);
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start2, stop2);
  std::cout << milliseconds << " elapsed slow" << std::endl;


  int* h_sorted = new int[array_size];
  hipMemcpy(h_sorted, d_sorted, sizeof(int) * array_size, hipMemcpyDeviceToHost);

  delete[] h_array;
  delete[] h_sorted;

}
